
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
using namespace std;

#define int unsigned int

class SparseMatrixInput{

public:
    int n;
    int m;
    int k;
    int new_n;
    int* block_data;
    int* block_index;
    int* cuda_block_data;
    int* cuda_block_index;

    SparseMatrixInput(){
        this->n = 0;
        this->m = 0;
        this->k = 0;
        this->new_n = 0;
        this->block_data = NULL;
        this->block_index = NULL;
    }

    void initialize_arrays(){
        this->new_n = this->n/this->m;
        this->block_data = new int[this->k*this->m*this->m];
        this->block_index = new int[(this->new_n)*(this->new_n)];
        // intialize the block index to -1
        for (int i=0;i<(this->new_n)*(this->new_n);i++){
            this->block_index[i] = this->n*this->n;
        }
        // copy the data to the gpu
        hipMalloc((void**)&this->cuda_block_data, this->k*this->m*this->m*sizeof(int));
        hipMalloc((void**)&this->cuda_block_index, (this->new_n)*(this->new_n)*sizeof(int));
    }

    void copy_to_cuda(){
        hipMemcpy(this->cuda_block_data, this->block_data, this->k*this->m*this->m*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(this->cuda_block_index, this->block_index, (this->new_n)*(this->new_n)*sizeof(int), hipMemcpyHostToDevice);
    }


    // print the entire matrix
    void print_matrix(){
        cout << "n: " << this->n << " m: " << this->m << " k: " << this->k << endl;
        for (int i=0;i<this->new_n*this->new_n;i++){
            if (this->block_index[i] == this->n*this->n)
                continue;
            cout << "Block index: " << i/this->new_n << " " << i%this->new_n << endl;
            for (int j=0;j<this->m*this->m;j++){
                cout << this->block_data[this->block_index[i]+j] << " ";
            }
            cout << endl;
        }
    }


    // make a destructor which deallocate memory from cuda as well
    ~SparseMatrixInput(){
        delete[] this->block_data;
        delete[] this->block_index;
        hipFree(this->cuda_block_data);
        hipFree(this->cuda_block_index);
    }


};


class SparseMatrixOutput{

public:
    int n;
    int m;
    int k;
    int new_n;
    int* block_data;
    int* block_index;
    int* cuda_block_data;
    int* cuda_block_index;

    SparseMatrixOutput(int n, int m){
        this->n = n;
        this->m = m;
        this->k = 0;
        this->new_n = n/m;
        this->block_data = new int[this->new_n*this->new_n*this->m*this->m];
        this->block_index = new int[this->new_n*this->new_n];
        // intialize the block index to -1
        for (int i=0;i<(this->new_n)*(this->new_n);i++){
            this->block_index[i] = i*m*m;
            // data values to zero 
            for (int j=0;j<this->m*this->m;j++){
                this->block_data[i*m*m+j] = 0;
            }
        }
        // copy the data to the gpu
        hipMalloc((void**)&this->cuda_block_data, this->new_n*this->new_n*this->m*this->m*sizeof(int));
        hipMalloc((void**)&this->cuda_block_index, (this->new_n)*(this->new_n)*sizeof(int));
        hipMemcpy(this->cuda_block_data, this->block_data, this->new_n*this->new_n*this->m*this->m*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(this->cuda_block_index, this->block_index, (this->new_n)*(this->new_n)*sizeof(int), hipMemcpyHostToDevice);
    }

    void check_zeros(){
        for (int i=0;i<this->new_n*this->new_n;i++){
            int flag = 0;
            for (int j=0;j<this->m*this->m;j++){
                if (this->block_data[this->block_index[i]+j] != 0){
                    flag = 1;
                    break;
                }
            }
            if (flag == 0){
                this->block_index[i] = this->n*this->n;
            }
            else{
                this->k++;
            }
        }
    }

    // print the entire matrix
    void print_matrix(){
        cout << "n: " << this->n << " m: " << this->m << " k: " << this->k << endl;
        for (int i=0;i<this->new_n*this->new_n;i++){
            if (this->block_index[i] == this->n*this->n)
                continue;
            cout << "Block index: " << i/this->new_n << " " << i%this->new_n << endl;
            for (int j=0;j<this->m*this->m;j++){
                cout << this->block_data[this->block_index[i]+j] << " ";
            }
            cout << endl;
        }
    }

    // make a destructor which deallocate memory from cuda as well
    ~SparseMatrixOutput(){
        delete[] this->block_data;
        delete[] this->block_index;
        hipFree(this->cuda_block_data);
        hipFree(this->cuda_block_index);
    }

};


// read 4 bytes of data from the input file and return the integer 
int readInt(ifstream &input){
    int result;
    input.read((char*)&result, sizeof(int));
    return result;
}


void read_input(SparseMatrixInput* matrix, ifstream &input){

    int n = matrix->n = readInt(input);
    int m = matrix->m = readInt(input);
    int k = matrix->k= readInt(input);
    matrix->initialize_arrays();
    int nn = n/m;

    int iter =0;
    for (int z=0;z<k;z++){
        int i = readInt(input);
        int j = readInt(input);
        // cout << i << " " << j << endl;
        matrix->block_index[i*nn + j] = iter;
        for (int a=0;a<m*m;a++){
            unsigned short val;
            input.read(( char *)&val, sizeof(unsigned short));
            matrix->block_data[iter++] = (int)val;
        }
    }
    // copy the data to the gpu
    matrix->copy_to_cuda();
}

void write_output(SparseMatrixOutput* result, ofstream &output){
    output.write((char*)&result->n, sizeof(int));
    output.write((char*)&result->m, sizeof(int));
    output.write((char*)&result->k, sizeof(int));

    for (int i=0; i<result->new_n*result->new_n; i++){
        if (result->block_index[i] != result->n*result->n){
            int ii = i/result->new_n;
            int jj = i%result->new_n;
            output.write((char*)&ii, sizeof(int));
            output.write((char*)&jj, sizeof(int));
            output.write((char*)&result->block_data[result->block_index[i]], sizeof(int)*result->m*result->m);
        }
    }
}

// multiply the two matrices and store the result in the result matrix using cuda 
__global__ void multiply(int* index1, int* index2, int* data1, int* data2, int* result_data, int n, int m){
    
    // printf("Hello\n");
    
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int nn = n/m;
    int mm = m*m;

    int i = index/nn;
    int j = index%nn;


    // cout << index << " " << i << " " << j << endl;

    if (index < nn*nn ){
        for (int idx = 0; idx < nn; idx++){
            int block1_index = index1[i*nn + idx];
            int block2_index = index2[idx*nn + j];
            // printf("%d %d %d %d\n",i ,j, block1_index, block2_index);
            if (block1_index != n*n && block2_index != n*n){
                // printf("hello\n");
                for (int a=0;a<m;a++){
                    for (int b=0;b<m;b++){
                        for (int c=0;c<m;c++){
                            unsigned long val = result_data[i*nn*mm + j*mm + a*m + b] + (unsigned long) data1[block1_index + a*m + c]*data2[block2_index + c*m + b];
                            result_data[i*nn*mm + j*mm + a*m + b] = min(val,(unsigned long) 0xffffffff);
                        }
                    }
                }
            }
        }
    }
}

void print_time(string s, chrono::high_resolution_clock::time_point& start, chrono::high_resolution_clock::time_point& end){
    if (s == "Total time"){
    end = chrono::high_resolution_clock::now();
    // print the time duration in mili seconds
    cout << s << " " << chrono::duration_cast<chrono::nanoseconds>(end - start).count()*1e-6 << " mseconds" << endl;
    start = chrono::high_resolution_clock::now();
    }
}


int32_t main( int32_t argc, char** argv ){
    if (argc !=4){
        cout <<"Wrong input format";
    }
    auto start_total = chrono::high_resolution_clock::now();
    auto start = chrono::high_resolution_clock::now();
    auto end = chrono::high_resolution_clock::now();

    string inputFile1 = argv[1];
    string inputFile2 = argv[2];
    string outputFile = argv[3];

    ifstream input1(inputFile1, ios::binary);
    ifstream input2(inputFile2, ios::binary);
    ofstream output(outputFile, ios::binary);

    print_time("Opening the input files", start, end);

    SparseMatrixInput* matrix1  = new SparseMatrixInput();
    read_input(matrix1, input1);

    SparseMatrixInput* matrix2  = new SparseMatrixInput();
    read_input(matrix2, input2);

    int n = matrix1->n;
    int m = matrix1->m;
    SparseMatrixOutput* result = new SparseMatrixOutput(n,m);

    print_time("Reading the input files", start, end);

    multiply<<<(n * n + 1023) / 1024, 1024>>>(matrix1->cuda_block_index, matrix2->cuda_block_index, matrix1->cuda_block_data, matrix2->cuda_block_data, result->cuda_block_data, n, m);

    // synchronize the threads
    hipDeviceSynchronize();

    print_time("multiplication", start, end);

    // copy the result back to the host
    hipMemcpy(result->block_data, result->cuda_block_data, result->new_n*result->new_n*result->m*result->m*sizeof(int), hipMemcpyDeviceToHost);
    result->check_zeros();
    write_output(result, output);

    print_time("writing output", start, end);

    print_time("Total time", start_total, end);

}